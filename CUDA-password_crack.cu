#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <time.h>

  struct  timespec start, end;
  long long int time_taken;

__device__ int is_a_match(char *password_crack)
{
  char password_number_1[] = "AA9996";
  char password_number_2[] = "AS1234";
  char password_number_3[] = "LK9091";
  char password_number_4[] = "FD1223";
  char *number_1 = password_crack;
  char *number_2 = password_crack;
  char *number_3 = password_crack;
  char *number_4 = password_crack;
  char *password_1 = password_number_1;
  char *password_2 = password_number_2;
  char *password_3 = password_number_3;
  char *password_4 = password_number_4;

  while(*number_1 == *password_1) 
  {
   if(*number_1 == '\0')
    {
    printf("***match found*** %s\n",password_number_1);
      break;
    }
    number_1++;
    password_1++;
  }
    
  while(*number_2 == *password_2)
  {
   if(*number_2 == '\0')
    {
    printf("***match found*** %s\n",password_number_2);
      break;
    }
    number_2++;
    password_2++;
  }

  while(*number_3 == *password_3) 
  {
   if(*number_3 == '\0')
    {
    printf("***match found*** %s\n",password_number_3);
      break;
    }
    number_3++;
    password_3++;
  }

  while(*number_4 == *password_4)
  {
   if(*number_4 == '\0')
    {
    printf("***match found*** %s\n",password_number_4);
      return 1;
    }
    number_4++;
    password_4++;
  }
  return 0;
}
__global__ void  kernel() 
{
  char a,b,c,d;
  char password[7];
  password[6] = '\0';
  int i = blockIdx.x+65;
  int j = threadIdx.x+65;
  char firstValue = i;
  char secondValue = j;
  password[0] = firstValue;
 password[1] = secondValue;
    for(a='0'; a<='9'; a++){
      for(b='0'; b<='9'; b++){
        for(c='0';c<='9';c++){
          for(d='0';d<='9';d++){
            password[2] = a;
            password[3] = b;
            password[4]= c;
            password[5]=d;
          if(is_a_match(password)) {
        //printf("***match found***");
          }
             else {
         //printf(" %s\n", password);          
            }
          }
        } 
      }
   }
}

int t_difference(struct timespec *start, struct timespec *end, long long int *difference) 
{
  long long int ds =  end->tv_sec - start->tv_sec;
  long long int dn =  end->tv_nsec - start->tv_nsec;

  if(dn < 0 ) 
  {
    ds--;
    dn += 1000000000;
  }
  *difference = ds * 1000000000 + dn;
  return !(*difference > 0);
}

int main() {
  clock_gettime(CLOCK_MONOTONIC, &start);
  kernel <<<26,26>>>();
  hipDeviceSynchronize();
  clock_gettime(CLOCK_MONOTONIC, &end);
  t_difference(&start, &end, &time_taken);
  printf("Total time taken was %lldns or %0.9lfs\n", time_taken, (time_taken/1.0e9));
  return 0;
}



